#include "hip/hip_runtime.h"
#include<sys/time.h>
#include<stdio.h>
#include<stdlib.h>
#include"bmp.h"
#include"function.h"
#include <hip/hip_runtime.h>
#include<malloc.h>
#define THREAD_NUM 512
int pBmpBuf_size;
int width; 
int height; //ͼ��߶�
int biClrUsed; //ʵ��ʹ�õ���ɫ��������
RGBQUAD *pColorTable; //��ɫ��ָ��
unsigned char *pBmpBuf; //ͼ������ָ��
int biBitCount; //ͼ������λ��
int BLOCK_NUM;


int readBmp(char *bmpName){
	FILE *fp; //�����ƴ��ļ�
	BITMAPINFOHEADER head; //λͼ��Ϣͷ
	int lineByte;
	
	fp=fopen(bmpName,"rb");
	//���ָ��δָ���κ�ͼ��
	if(fp==NULL){
		return 0;
	}
	//����λͼ�ļ��ļ�ͷ
	fseek(fp,sizeof(BITMAPFILEHEADER),0);	//��ָ���ƶ������뿪ʼ��СΪBITMAPFILEHEADER�ֽڴ�
	printf("1..%ld\n",ftell(fp));

	fread(&head,sizeof(BITMAPINFOHEADER),1,fp);
	printf("size unsigned int%ld,\n",sizeof(unsigned int));
	printf("sizeof%ld,%ld\n",sizeof(BITMAPFILEHEADER),sizeof(BITMAPINFOHEADER));
	printf("%ld,%ld,%ld,%ld\n",sizeof(head.biWidth),sizeof(head.biHeight),sizeof(head.biBitCount),sizeof(head.biClrUsed));
	printf("2..%ld\n",ftell(fp));
	//ͨ��head��ȡͼ���ȣ��߶ȵ���Ϣ
	width=head.biWidth;		//ͼ��Ŀ�
	height=head.biHeight;	//ͼ��ĸ�
	biBitCount=head.biBitCount;		//����λ��
	biClrUsed=head.biClrUsed;		//ʵ��ʹ�õ���ɫ��
	//����ͼ��ÿ��ռ��������
	lineByte=(width*biBitCount/8+3)/4*4;	//ע��λͼÿ��ռ���ֽ���������4�ı�����+3���ã�
	
	//���Ҫ��ȡ���ǻҶ�ͼ��,��Ҫ��ȡ��ɫ��
	if(biBitCount==8){
		pColorTable=(RGBQUAD*)malloc(sizeof( RGBQUAD)*256);
		fread(pColorTable,sizeof(RGBQUAD),256,fp); //��ʱfp���������ļ����λ��
	}
	
	//��ȡλͼͼ������
	pBmpBuf=(unsigned char*)malloc(sizeof(unsigned char)*(lineByte*height));
	pBmpBuf_size=lineByte*height;
	fread(pBmpBuf,1,lineByte*height,fp); //ÿ�ζ�һ���ֽ�
	fclose(fp); //�ر��ļ�
	
	return 1;
}

/*******************************************************************
*	��������saveBmp()                                              *
*	���ܣ�����Ҷ�ͼ                                               *
*	����������                                                     *
*	char *bmpName //����Ҷ�ͼ����-                                *
*	unsigned char *bmpBuf //λͼ����ָ�룬�����ж��Ƿ������ݴ���   *
*	int pWidth //λͼ��                                            *
*	int pHeight //λͼ��                                           *
*	int biBitCount //λͼ����λ��                                  *
*	RGBQUAD *pColorTable //��ɫ��ָ��                              *
*******************************************************************/
int saveBmp(char *bmpName,unsigned char *bmpBuf,int pWidth,int pHeight,int biBigCount,RGBQUAD *pColorTable){
	//��������ݴ���
	if(bmpBuf==NULL){
		return 0;
	}
	int colorTableSize=0;//������ɫ���С
	//���Ҫ������ǻҶ�ͼ
	if(biBitCount==8){
		colorTableSize=1024;//�Ҷ�ͼ����ɫ���СΪ256*4=1024
	}
	int lineByte=(pWidth*biBitCount/8+3)/4*4;//ÿ��ռ�ݵ��ֽ���
	//�����������ļ�
	FILE *fp=fopen(bmpName,"wb");
	//��������ļ�ʧ��
	if(fp==NULL){
		printf("Creat file failed !");
		return 0;
	}
	//����λͼͼ���ļ�ͷ
	BITMAPFILEHEADER filehead;//λͼ�ļ��ļ�ͷ
	filehead.bfType=0x4D42; //bmp λͼ��ʽ
	filehead.bfSize=sizeof(BITMAPFILEHEADER)+sizeof(BITMAPINFOHEADER)+colorTableSize+lineByte*pHeight;
	filehead.bfReserved1=0;
	filehead.bfReserved2=0;
	//bfOffBits ��ֵ��λͼ�ļ�ǰ����������ռ�֮��
	filehead.bfOffBits=54+colorTableSize;
	//���ļ�ͷд���ļ���
	fwrite(&filehead,sizeof(BITMAPFILEHEADER),1,fp);
	
	//����λͼ��Ϣͷ
	BITMAPINFOHEADER infohead;
	//������Ϣͷ��ز���
	infohead.biBitCount=biBitCount; //����ͼ������λ��
	infohead.biClrImportant=0; //ͼ����ɫ��������Ҫ
	infohead.biClrUsed=0; //ʹ����ɫ���е�������ɫ����
	infohead.biCompression=0; //ͼ��ѹ������
	infohead.biHeight=pHeight; //ͼ��߶�
	infohead.biPlanes=1; //ΪĿ���豸˵��λ��������ֵ�����Ǳ���Ϊ1
	infohead.biSize=40; //BITMAPINFOHEADER�ṹ����Ҫ������
	infohead.biSizeImage=lineByte*pHeight; //ͼ���С
	infohead.biWidth=pWidth;
	infohead.biXPelsPerMeter=0; //ˮƽ�ֱ���
	infohead.biYPelsPerMeter=0; //��ֱ�ֱ���
	
	//��λͼ��Ϣͷд���ļ�
	fwrite(&infohead,sizeof(BITMAPINFOHEADER),1,fp);
	
	//����ǻҶ�ͼ��Ҫ���Ҷ�ͼ����ɫ��Ҳд���ļ��ſ���
	if(biBitCount==8){
		fwrite(pColorTable,sizeof(RGBQUAD),256,fp);
	}
	//��λͼ����д���ļ�
	fwrite(bmpBuf,lineByte*pHeight,1,fp); //һ��дһ���ֽ�
	fclose(fp);
	
	return 1;
}


__global__ void gray_f(unsigned char *pBmpBuf,int *lineByte,int *height,int *width)
{
	int gray;
	int i,j;
	BYTE red,green,blue;
	int tid=threadIdx.x;
	int bid=blockIdx.x;
	i=bid*THREAD_NUM+tid;
	if(i<*height)
	{
		for(j=0;j<*width;j++)
		{
			blue=*(pBmpBuf+i*(*lineByte)+j*3+0);
			green=*(pBmpBuf+i*(*lineByte)+j*3+1);
			red=*(pBmpBuf+i*(*lineByte)+j*3+2);
			gray=(30*red+59*green+11*blue)/100;
			*(pBmpBuf+i*(*lineByte)+j*3+0)=(BYTE)gray; 
			*(pBmpBuf+i*(*lineByte)+j*3+1)=(BYTE)gray; 
			*(pBmpBuf+i*(*lineByte)+j*3+2)=(BYTE)gray; 
		}
	}
}

int main(){
	struct timeval t1,t2;
	double timeuse;
	gettimeofday(&t1,NULL);
	char readPath[]="10000_7500.bmp"; //����ͼ������
	readBmp(readPath); //����ͼ��
	int gray; //�Ҷ�ֵ
	
	printf("the width: %d\nthe height: %d\nthe biBitCount: %d\n",width,height,biBitCount);
	printf("the biclrused: %d\n",biClrUsed);
	int i;
	int lineByte=(width*biBitCount/8+3)/4*4;
	int *a;
	unsigned char *b;
	int *c,*d;
	hipMalloc(&a,sizeof(int));
	hipMalloc(&c,sizeof(int));
	hipMalloc(&d,sizeof(int));
	hipMalloc(&b,sizeof(unsigned char)*pBmpBuf_size);
	hipMemcpy(a,&lineByte,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(c,&height,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d,&width,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(b,pBmpBuf,sizeof(unsigned char)*pBmpBuf_size,hipMemcpyHostToDevice);
	if(biBitCount==8){}
	else if(biBitCount==24){
				BLOCK_NUM=width/THREAD_NUM+1;
				gray_f<<<BLOCK_NUM,THREAD_NUM>>>(b,a,c,d);
				hipMemcpy(pBmpBuf,b,sizeof(unsigned char)*pBmpBuf_size,hipMemcpyDeviceToHost);
			}
		
	hipFree(a);
	hipFree(b);	
	hipFree(c);	
	hipFree(d);	
	//�����256ɫͼ��
	if(biBitCount==8){
		for(i=0;i<256;i++){
			gray=(11*pColorTable[i].rgbBlue+30*pColorTable[i].rgbRed+59*pColorTable[i].rgbGreen)/100; //����Ҷ�ֵ
			pColorTable[i].rgbBlue=(BYTE)gray; //����ɫ�����滻Ϊ�Ҷ�ֵ
			pColorTable[i].rgbRed=(BYTE)gray; //����ɫ�����滻Ϊ�Ҷ�ֵ
			pColorTable[i].rgbGreen=(BYTE)gray; //����ɫ�����滻Ϊ�Ҷ�ֵ
		}
	}
	
	//����Ҷ�ͼ��
	char writePath[]="10000_7500cu.bmp";
	saveBmp(writePath,pBmpBuf,width,height,biBitCount,pColorTable);//�洢�Ҷ�ͼ��
	//���new������ɵ�ȫ�ֱ���
	free(pBmpBuf);
	//����ǻҶ�ͼ
	if(biBitCount==8)
		free(pColorTable);
	gettimeofday(&t2,NULL);
	timeuse=t2.tv_sec-t1.tv_sec+(t2.tv_usec-t1.tv_usec)/1000000.0;
	printf("\ntime:%f\n",timeuse);
	return 0;	
}
	
	
