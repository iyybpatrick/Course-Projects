#include "hip/hip_runtime.h"
#include "function.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <stdbool.h>
#define WIDTHBYTES(bits) (((bits)+31)/32*4)
#pragma warning(disable:4996)

#define TRUE 1
#define FALSE 0

__global__ void update(BYTE *NewpGryImg, long *LineBytes,long *height, long *width, BYTE *pGryImg, float *tempArray)
{
	long id = threadIdx.x;
	id += (blockIdx.x * 512);
	if(id >= 0 && id < (((*height) - 3 + 1)*((*width) - 3 + 1)))
	{
		long temp = (*width) - 3 + 1;
		long id_i = id / temp;
		id_i += 1;
		long id_j = id % temp;
		id_j += 1;
		BYTE *lpSrc;
		BYTE *lpDst;
		lpDst = NewpGryImg + (*LineBytes)*((*height) - 1 - id_i) + id_j;
		float result = 0;
		for (long k = 0; k < 3; k++)
		{
			for (long l = 0; l < 3; l++)
			{
				lpSrc = pGryImg + (*LineBytes)*((*height) - 1 - id_i + 1 - k) + id_j - 1 + l;
				result += (*lpSrc)*tempArray[k*3 + l];
			}
		}
		result = result*(1.0);
		result = (float)fabs(result);
		if (result > 255)
		{
			*lpDst = 255;
		}
		else
		{
			*lpDst = (unsigned char)(result + 0.5);
		}
	}
}

bool Sobel(BYTE *pGryImg, int width, int height, int threshold);

BYTE *Read8BitBmpFile2Img(const char* filename, int *width, int *height)
{
	FILE *BinFile;
	BITMAPFILEHEADER FileHeader;
	BITMAPINFOHEADER BmpHeader;
	BYTE *plmg;
	unsigned int size;
	int Suc = 1, w, h;

	//open file
	*width = *height = 0;
	if ((BinFile = fopen(filename, "rb")) == NULL) return NULL;
	//read struct info
	if (fread((void*)&FileHeader, 1, sizeof(FileHeader), BinFile) != sizeof(FileHeader)) Suc = -1;
	if (fread((void*)&BmpHeader, 1, sizeof(BmpHeader), BinFile) != sizeof(BmpHeader)) Suc = -1;
	if ((Suc == -1) || (FileHeader.bfOffBits<sizeof(FileHeader)+sizeof(BmpHeader)))
	{
		fclose(BinFile);
		return NULL;
	}
	//read Image data
	*width = w = BmpHeader.biWidth;
	*height = h = BmpHeader.biHeight;
	size = w*h;
	fseek(BinFile, FileHeader.bfOffBits, SEEK_SET);
	if ((plmg = (BYTE*)malloc(sizeof(BYTE)*size)) != NULL)
	{
		for (int i = 0; i<h; i++)  //0,1,2,3,4(5);400-499
		{
			if (fread(plmg + (h - 1 - i)*w, sizeof(BYTE), w, BinFile) != w)
			{
				fclose(BinFile);
				free( plmg);
				plmg = NULL;
				return NULL;
			}
			fseek(BinFile, (w + 3) / 4 * 4 - w, SEEK_CUR);
		}
	}
	fclose(BinFile);
	return plmg;
}


bool Write8BitImg2BmpFile(BYTE *pImg, int width, int height, const char* filename)
//����Ȳ���4�ı���ʱ�Զ���ӳ�4�ı���
{
	FILE *BinFile;
	BITMAPFILEHEADER FileHeader;
	BITMAPINFOHEADER BmpHeader;
	int i, extend;
	bool Suc = true;
	BYTE p[4], *pCur;

	//Open File
	if ((BinFile = fopen(filename, "w+b")) == NULL) { return false; }
	//Fill the FileHeader
	FileHeader.bfType = ((WORD)('M' << 8) | 'B');
	FileHeader.bfOffBits = sizeof(BITMAPFILEHEADER)+sizeof(BITMAPINFOHEADER)+256 * 4L;
	FileHeader.bfSize = FileHeader.bfOffBits + width*height;
	FileHeader.bfReserved1 = 0;
	FileHeader.bfReserved2 = 0;
	if (fwrite((void*)&FileHeader, 1, sizeof(FileHeader), BinFile) != sizeof(FileHeader)) Suc = false;
	//Fill the ImgHeader
	BmpHeader.biSize = 40;
	BmpHeader.biWidth = width;
	BmpHeader.biHeight = height;
	BmpHeader.biPlanes = 1;
	BmpHeader.biBitCount = 8;
	BmpHeader.biCompression = 0;
	BmpHeader.biSizeImage = 0;
	BmpHeader.biXPelsPerMeter = 0;
	BmpHeader.biYPelsPerMeter = 0;
	BmpHeader.biClrUsed = 0;
	BmpHeader.biClrImportant = 0;
	if (fwrite((void*)&BmpHeader, 1, sizeof(BmpHeader), BinFile) != sizeof(BmpHeader)) Suc = false;
	//write Pallete
	for (i = 0, p[3] = 0; i<256; i++)
	{
		p[3] = 0;
		p[0] = p[1] = p[2] = i;//blue,green,red
		if (fwrite((void*)p, 1, 4, BinFile) != 4) { Suc = false; break; }
	}
	//write image data
	extend = (width + 3) / 4 * 4 - width;
	if (extend == 0)
	{
		for (pCur = pImg + (height - 1)*width; pCur >= pImg; pCur -= width)
		{
			if (fwrite((void*)pCur, 1, width, BinFile) != (unsigned int)width) Suc = false;//��ʵ������
		}
	}
	else
	{
		for (pCur = pImg + (height - 1)*width; pCur >= pImg; pCur -= width)
		{
			if (fwrite((void*)pCur, 1, width, BinFile) != (unsigned int)width) Suc = false;//��ʵ������
			for (i = 0; i<extend; i++)//���������
			if (fwrite((void*)(pCur + width - 1), 1, 1, BinFile) != 1) Suc = false;
		}
	}
	//return
	fclose(BinFile);
	return Suc;
}

bool Template(BYTE *pGryImg, long width, long height, float * tempArray)
{
	long LineBytes;
	LineBytes = WIDTHBYTES(width * 8);
	//prepare for cuda
	long *LineBytesGPU;
	long *heightGPU;
	long *widthGPU;
	BYTE *NewpGryImgGPU;
	BYTE *pGryImgGPU;
	float *tempArrayGPU;
	hipMalloc(&LineBytesGPU, sizeof(long));
	hipMalloc(&heightGPU, sizeof(long));
	hipMalloc(&widthGPU, sizeof(long));
	hipMalloc(&tempArrayGPU, sizeof(float)*9);
	hipMalloc(&NewpGryImgGPU, sizeof(BYTE)*(LineBytes*height));
	hipMalloc(&pGryImgGPU, sizeof(BYTE)*(LineBytes*height));
	hipMemcpy(LineBytesGPU, &LineBytes, sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(heightGPU, &height, sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(widthGPU, &width, sizeof(long), hipMemcpyHostToDevice);
	hipMemcpy(tempArrayGPU, tempArray, sizeof(float)*9, hipMemcpyHostToDevice);
	hipMemcpy(pGryImgGPU, pGryImg, sizeof(BYTE)*(LineBytes*height), hipMemcpyHostToDevice);
	//kernel proce
	int blockPerGird = ((height - 3 + 1)*(width - 3 + 1)/512) + 1;
	update<<<blockPerGird,512>>>(NewpGryImgGPU, LineBytesGPU, heightGPU, widthGPU, pGryImgGPU, tempArrayGPU);
	hipDeviceSynchronize();
	//prepare for cpu op
	hipMemcpy(pGryImg, NewpGryImgGPU, (LineBytes*height)*sizeof(BYTE), hipMemcpyDeviceToHost);
	//memcpy(pGryImg, NewpGryImg, LineBytes*height);
	return TRUE;
}

bool Sobel(BYTE *pGryImg, int width, int height, int threshold)
{
	BYTE *lpDst1;
	BYTE *lpDst2;
	BYTE *NewpGryImg1;
	BYTE *hNewGryImg1;
	BYTE *NewpGryImg2;
	BYTE *hNewGryImg2;
	long i, j;
	float aTemplate[9];

	hNewGryImg1 = (BYTE*)malloc(sizeof(BYTE)*width * height);
	if (hNewGryImg1 == NULL)
	{
		return FALSE;
	}
	NewpGryImg1 = hNewGryImg1;
	hNewGryImg2 = (BYTE*)malloc(sizeof(BYTE)*width * height);
	if (hNewGryImg2 == NULL)
	{
		return FALSE;
	}
	NewpGryImg2 = hNewGryImg2;
	lpDst1 = NewpGryImg1;
	memcpy(NewpGryImg1, pGryImg, width * height);
	lpDst2 = NewpGryImg2;
	memcpy(NewpGryImg2, pGryImg, width * height);
	// ����Sobelģ�����
	aTemplate[0] = -1.0;
	aTemplate[1] = -2.0;
	aTemplate[2] = -1.0;
	aTemplate[3] = 0.0;
	aTemplate[4] = 0.0;
	aTemplate[5] = 0.0;
	aTemplate[6] = 1.0;
	aTemplate[7] = 2.0;
	aTemplate[8] = 1.0;
	if (!Template(NewpGryImg1, width, height, aTemplate))
	{
		return FALSE;
	}
	// ����Sobelģ�����
	aTemplate[0] = -1.0;
	aTemplate[1] = 0.0;
	aTemplate[2] = 1.0;
	aTemplate[3] = -2.0;
	aTemplate[4] = 0.0;
	aTemplate[5] = 2.0;
	aTemplate[6] = -1.0;
	aTemplate[7] = 0.0;
	aTemplate[8] = 1.0;
	if (!Template(NewpGryImg2, width, height, aTemplate))
	{
		return FALSE;
	}
	for (j = 0; j <height; j++)
	{
		for (i = 0; i <width - 1; i++)
		{
			lpDst1 = NewpGryImg1 + width * j + i;
			lpDst2 = NewpGryImg2 + width * j + i;
			if (*lpDst2 > *lpDst1)
			{
				*lpDst1 = *lpDst2;
			}
			if (*lpDst1>threshold)
			{
				*lpDst1 = 255;
			}
			if (*lpDst1<threshold)
			{
				*lpDst1 = 0;
			}
		}
	}
	memcpy(pGryImg, NewpGryImg1, width * height);
	return TRUE;
}




int main()
{
	clock_t start, finish;
	int width, height, threshold = 0;
	start = clock();
	BYTE *pGryImg = Read8BitBmpFile2Img("lena.bmp", &width, &height);
	BYTE *dstImg = (BYTE*) malloc(sizeof(BYTE)*width*height);
	//printf("%d,%d\n",width,height);  
	printf("Robert���ӱ�Ե���\n");
	//printf("��������ֵ:");
	//scanf("%d", &threshold);
	threshold = 50;
	//pGryImg = Read8BitBmpFile2Img("lena.bmp", &width, &height);
	Sobel(pGryImg, width, height, threshold);
	Write8BitImg2BmpFile(pGryImg, width, height, "lena_Sobel.bmp");
	free( pGryImg);
	finish = clock();
	float time = 0;
	printf("time:%f\n", time = (float)(finish - start / CLOCKS_PER_SEC));

	return 0;
}
