/**********************************************************************
* DESCRIPTION:
* Serial Concurrent Wave Equation - C Version
* This program implements the concurrent wave equation
*********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define THREAD_NUM 512
#define BLOCK_NUM 64
void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);
int nsteps , /* number of time steps */
tpoints , /* total points along string */
rcode; /* generic return code */
float values[MAXPOINTS+2], /* values at time t */
oldval[MAXPOINTS+2], /* values at time (t-dt) */
newval[MAXPOINTS+2]; /* values at time (t+dt) */
/**********************************************************************
* Checks input values from parameters
*********************************************************************/
void check_param(void)
{
char tchar[20];
/* check number of points , number of iterations */
while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
printf("Enter number of points along vibrating string [%d-%d]: "
,MINPOINTS , MAXPOINTS);
scanf("%s", tchar);
tpoints = atoi(tchar);
if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
printf("Invalid. Please enter value between %d and %d\n",
MINPOINTS , MAXPOINTS);
}
while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
scanf("%s", tchar);
nsteps = atoi(tchar);
if ((nsteps < 1) || (nsteps > MAXSTEPS))
printf("Invalid. Please enter value between 1 and %d\n",
MAXSTEPS);
}
printf("Using points = %d, steps = %d\n", tpoints , nsteps);
}
/**********************************************************************
* Initialize points on line
*********************************************************************/
void init_line(void)
{
int i, j;
float x, fac, k, tmp;
/* Calculate initial values based on sine curve */
fac = 2.0 * PI;
k = 0.0;
tmp = tpoints - 1;
for (j = 1; j <= tpoints; j++) {
x = k/tmp;
values[j] = sin (fac * x);
k = k + 1.0;
}
/* Initialize old values array */
for (i = 1; i <= tpoints; i++)
oldval[i] = values[i];
}
/**********************************************************************
* Calculate new values using wave equation
*********************************************************************/
__global__ static void do_math(float *oldval, float*values, float *newval, int *nsteps1, int *tpoints1)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int i, j;

        int nsteps=nsteps1[0];
        int tpoints=tpoints1[0];
       /* 
        float *oldval = new float[tpoints];
	float *values = new float[tpoints];
	float *newval = new float[tpoints];

	oldval = oldval1;
	values = values1;
	newval = newval1;
       */
	float dtime, c, dx, tau, sqtau;
	dtime = 0.3;
	c = 1.0;
	dx = 1.0;
	tau = (c * dtime / dx);
	sqtau = tau * tau;

	for (i = 1; i <= nsteps; i++) {
		for (j = bid*THREAD_NUM + tid; j <= tpoints; j += BLOCK_NUM*THREAD_NUM)
		{

			newval[j] = (2.0 * values[j]) - oldval[j] + (sqtau * (-2.0)*values[
				j]);
				oldval[j] = values[j];
				values[j] = newval[j];
		}
		if (tid == 0)
		{
			values[0] = 0.0;
			newval[0] = 0.0;
		}
		if (tid == BLOCK_NUM)
		{
			values[tpoints] = 0.0;
			newval[tpoints] = 0.0;
		}
            	//	__syncthreads();
	}

}

/**********************************************************************
* Print final results
*********************************************************************/
void printfinal()
{
int i;
for (i = 1; i <= tpoints; i++) {
printf("%6.4f ", values[i]);
if (i%10 == 0)
printf("\n");
}
}
/**********************************************************************
* Main program
*********************************************************************/
int main(int argc, char *argv[])
{

     //	clock_t start, finish;
        
	sscanf(argv[1], "%d", &tpoints);
	sscanf(argv[2], "%d", &nsteps);
	check_param();
	printf("Initializing points on the line...\n");
	init_line();
	printf("Updating all points for all time steps...\n");

     //	start = clock();
	int DATA_SIZE = tpoints;

	/*Update function*/
	int *gpu_oldval, *gpu_values, *gpu_newval,*nsteps1,*tpoints1;
	hipMalloc((void**)&gpu_oldval, sizeof(float)*DATA_SIZE);
	hipMalloc((void**)&gpu_values, sizeof(float)*DATA_SIZE);
	hipMalloc((void**)&gpu_newval, sizeof(float)*DATA_SIZE);
        hipMalloc((void**)&nsteps1,sizeof(int)*1);
        hipMalloc((void**)&tpoints1,sizeof(int)*1);


	hipMemcpy(gpu_oldval, oldval, sizeof(float)*DATA_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(gpu_values, values, sizeof(float)*DATA_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(gpu_newval, newval, sizeof(float)*DATA_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(nsteps1,&nsteps,sizeof(int)*1,hipMemcpyHostToDevice);
        hipMemcpy(tpoints1,&tpoints,sizeof(int)*1,hipMemcpyHostToDevice);
        

	do_math <<< BLOCK_NUM, THREAD_NUM >>>((float*)gpu_oldval,(float*) gpu_values,(float*) gpu_newval,nsteps1,tpoints1);

	hipMemcpy(values, gpu_values, sizeof(float)*DATA_SIZE, hipMemcpyDeviceToHost);


	hipFree(gpu_oldval);
	hipFree(gpu_values);
	hipFree(gpu_newval);


	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
     //	finish = clock();
     //	printf("time:%f\n", (float)(finish - start) /( CLOCKS_PER_SEC));
	return 0;
}
